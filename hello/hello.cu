#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

//using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
//rtBuffer<float4, 2>   result_buffer;
__device__ optix::buffer<float4, 2> result_buffer;

rtDeclareVariable(float3, draw_color, , );

RT_PROGRAM void hello()
{
  //result_buffer[launch_index] = make_float4(draw_color, 0.f);
	result_buffer[launch_index] = make_float4(0.25, 0.36, 0.29, 0.f);
}



/*
#include <optix.h>


using namespace optix;
//RTbuffer result_buffer;
//__device__ optix::buffer<float4, 1> result_buffer;
rtBuffer<float4, 2>   result_buffer;

//rtDeclareVariable(float3,                draw_color, , );

//RT_PROGRAM void draw_solid_color()
//{
 // result_buffer[launch_index] = make_float4(draw_color, 0.f);
//}

RT_PROGRAM void hello()
{
  result_buffer[0] = make_float4(1, 1, 0, 0.f);
}
*/

/*
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2>   result_buffer;

rtDeclareVariable(float3,                draw_color, , );

RT_PROGRAM void draw_solid_color()
{
  result_buffer[launch_index] = make_float4(draw_color, 0.f);
}

*/