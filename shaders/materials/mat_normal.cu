#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_math.h>
//#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float3, prd_color, rtPayload, );

RT_PROGRAM void closest_hit() {
	prd_color = normalize(normal)*0.5f+0.5f;
}