#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_math.h>
#include "common.cuh"
//#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(float3, normal, attribute normal, );

RT_PROGRAM void closest_hit() {

	prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normal))*0.5f + 0.5f;
	//printf("closest_hit");
}