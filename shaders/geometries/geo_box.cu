#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

rtDeclareVariable(float3, box_min, , );
rtDeclareVariable(float3, box_max, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, normal, attribute normal, );

static __device__ float3 box_normal(float t, float3 d0, float3 d1) {

	float3 neg = make_float3(t==d0.x?1:0, t==d0.y?1:0, t==d0.z?1:0);
	float3 pos = make_float3(t==d1.x?1:0, t==d1.y?1:0, t==d1.z?1:0);

	return pos-neg;

}

RT_PROGRAM void box_intersect(int) {

	float3 d0 = (box_min-ray.origin)/ray.direction;
	float3 d1 = (box_max-ray.origin)/ray.direction;

	float3 dmin = fminf(d0, d1);
	float3 dmax = fmaxf(d0, d1);
	float t0 = fmaxf(dmin);
	float t1 = fminf(dmax);
	if (t0 <= t1) {
		bool check_second = true;
		if (rtPotentialIntersection(t0)) {
			normal = box_normal(t0, d0, d1);
			if (rtReportIntersection(0)) {
				check_second = false;
			}
		}
		if (check_second) {
			if (rtPotentialIntersection(t1)) {
				normal = box_normal(t1, d0, d1);
				rtReportIntersection(0);
			}
		}
	}

}

RT_PROGRAM void box_bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->set(box_min, box_max);
}