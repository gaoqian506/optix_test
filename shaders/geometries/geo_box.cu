#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

rtDeclareVariable(float3, box_min, , );
rtDeclareVariable(float3, box_max, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, normal, attribute normal, );

static __device__ float3 box_normal(float t, float3 d0, float3 d1) {

	float3 neg = make_float3(t==d0.x?1:0, t==d0.y?1:0, t==d0.z?1:0);
	float3 pos = make_float3(t==d1.x?1:0, t==d1.y?1:0, t==d1.z?1:0);

	return pos-neg;

}

RT_PROGRAM void intersect(int) {

	float3 c0 = box_min-ray.origin;
	float3 c1 = box_max-ray.origin;
	float3 d0 = c0/ray.direction;
	float3 d1 = c1/ray.direction;

	// printf("c0:%f %f %f\nc1:%f %f %f\n", 
	// 		c0.x, c0.y, c0.z,
	// 		c1.x, c1.y, c1.z);


	// printf("box_min:%f %f %f, box_max:%f, %f, %f\n", 
	// 	box_min.x, box_min.y, box_min.z,
	// 	box_max.x, box_max.y, box_max.z);

	// printf("ray.origin:%f %f %f\n", 
	// 	ray.origin.x, ray.origin.y, ray.origin.z);
	// printf("ray.direction:%f %f %f\n", 
	// 	ray.direction.x, ray.direction.y, ray.direction.z);	

	//printf("d0:%f %f %f\n", d0.x, d0.y, d0.z);		

	float3 dmin = fminf(d0, d1);
	float3 dmax = fmaxf(d0, d1);
	float t0 = fmaxf(dmin);
	float t1 = fminf(dmax);
	if (t0 <= t1) {
		bool check_second = true;
		if (rtPotentialIntersection(t0)) {
			//printf("rtPotentialIntersection0");
			normal = box_normal(t0, d0, d1);
			if (rtReportIntersection(0)) {
				check_second = false;
			}
		}
		if (check_second) {
			if (rtPotentialIntersection(t1)) {
				normal = box_normal(t1, d0, d1);
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void bounds(int, float result[6]) {
	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->set(box_min, box_max);
}