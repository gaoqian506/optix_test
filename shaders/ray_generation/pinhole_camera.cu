#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_math.h>
//#include "common.cuh"

rtBuffer<float4, 2> output_buffer;


rtDeclareVariable(float3, eye, , );
//rtDeclareVariable(float3, roll_pitch_yaw, , );
//rtDeclareVariable(float3, fovy_cx_cy, , );
rtDeclareVariable(float3, target, , );
rtDeclareVariable(float, roll_angle, , );
rtDeclareVariable(float3, intrisics, , ); // [fovy, cx, cy]



rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


RT_PROGRAM void pinhole_camera() {

	optix::size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen);
	float3 dir = make_float3(d, 1)-0.5;
	optix::Ray ray(eye, dir, 0, 0.01);

	output_buffer[launch_index] = make_float4(d, 0, 1);
}


