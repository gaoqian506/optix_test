#include "hip/hip_runtime.h"

#include <optix.h>
#include "common.cuh"


rtDeclareVariable(float3, eye, , );
//rtDeclareVariable(float3, roll_pitch_yaw, , );
//rtDeclareVariable(float3, fovy_cx_cy, , );
rtDeclareVariable(float3, target, , );
rtDeclareVariable(float, roll_angle, , );
rtDeclareVariable(float3, intrisics, , ); // [fovy, cx, cy]



rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


RT_PROGRAM void pinhole_camera() {

	//output_buffer[launch_index] = make_float4(1, 1, 0, 1);
}


