#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_math.h>
//#include "common.cuh"

struct PerRayData_radiance
{
	float3 result;
	float  importance;
	int depth;
};

rtBuffer<float4, 2> output_buffer;


rtDeclareVariable(float3, eye, , );
//rtDeclareVariable(float3, roll_pitch_yaw, , );
//rtDeclareVariable(float3, fovy_cx_cy, , );
rtDeclareVariable(float3, target, , );
rtDeclareVariable(float, roll_angle, , );
rtDeclareVariable(float3, intrisics, , ); // [fovy, cx, cy]

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(rtObject, top_object, , );



rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );


RT_PROGRAM void pinhole_camera() {

	optix::size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen);
	float3 dir = make_float3(d, 1)-0.5;
	optix::Ray ray(eye, dir, 0, 0.01);

	PerRayData_radiance prd;
	prd.result = make_float3(1, 0, 0);

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_float4(prd.result, 1);

	//rtPrintf("hello");
}


