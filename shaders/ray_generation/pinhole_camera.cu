#include "hip/hip_runtime.h"

#include <optix.h>
#include <optix_math.h>
#include "common.cuh"




rtDeclareVariable(float3, eye, , );
//rtDeclareVariable(float3, roll_pitch_yaw, , );
//rtDeclareVariable(float3, fovy_cx_cy, , );
rtDeclareVariable(float3, target, , );
rtDeclareVariable(float, roll_angle, , );
rtDeclareVariable(float3, intrisics, , ); // [fovy, cx, cy]





RT_PROGRAM void pinhole_camera() {

	optix::size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen);
	float3 dir = make_float3(d-0.5, 1);
	optix::Ray ray(eye, dir, 0, 0.01);

	PerRayData_radiance prd;
	prd.result = make_float3(1, 0, 0);

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_float4(prd.result, 1);

	//rtPrintf("hello");
}


