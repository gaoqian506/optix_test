#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>


rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float3, prd_color, rtPayload, );
rtDeclareVariable(rtObject, top_object, , );
__device__ optix::buffer<float4, 2> result_buffer;


RT_PROGRAM void box()
{
	optix::Ray ray(make_float3(0, 0, 0), make_float3(0, 0, 1), 0, 0.1);

	float3 color = make_float3(1, 1, 0);
	//rtTrace(rtObject(), ray, color);
	result_buffer[launch_index] = make_float4(color, 1);
}

RT_PROGRAM void miss() {
	//prd_color = make_float3(1, 0, 0);
}

RT_PROGRAM void exception() {
	//result_buffer[launch_index] = make_float4(0, 0, 1, 1);
}

RT_PROGRAM void intersect(int) {

}

RT_PROGRAM void bounds(int, float result[6]) {

}

RT_PROGRAM void closest_hit() {
	//prd_color = make_float3(0, 1, 0);
}

// RT_PROGRAM void any_hit() {
// 	prd_color = make_float3(1, 1, 0);
// }
