#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>


rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
__device__ optix::buffer<uchar4, 2> result_buffer;

RT_PROGRAM void box()
{
  	result_buffer[launch_index] = make_uchar4(0, 255, 255, 255);
}